#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>

namespace testapi
{
	__global__ void fooTest()
	{
		printf("Hello world from the GPU!\n");
		printf("fooTest is here!");
	}


	void callFooKernelTest()
	{
		fooTest << <1, 1 >> > ();
	}
}